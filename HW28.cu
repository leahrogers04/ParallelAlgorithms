#include "hip/hip_runtime.h"
// Name:
// CPU random walk. 
// nvcc HW28.cu -o temp - lcurand

/*
 What to do:
 This is some code that runs a random walk for 10000 steps.
 Use cudaRand and run 10 of these runs at once with diferent seeds on the GPU.
 Print out all 10 final positions.
*/

// Include files
#include <sys/time.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

// Defines
#define WALKS 10
#define STEPS 10000
// Globals
//int NumberOfRandomSteps = 10000;
//float MidPoint = (float)RAND_MAX/2.0f;

// Function prototypes
//int getRandomDirection();
//__global__ void randomWalk(int, unsigned long long, int, float);
//int main(int, char**);

__global__ void randomWalk(int *finalPositions, unsigned long long seed, int numberOfRandomSteps, float midPoint)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id < WALKS)
	{
		hiprandState state;
		hiprand_init(seed + id, 0, 0, &state);

		int initialPosition = 0;

		for (int i = 0; i < numberOfRandomSteps; i++)
		{
			unsigned int randomNumber = hiprand(&state);
			int step = (randomNumber < midPoint) ? -1 : 1; //hiprand() returns a random number in [0, 1)
			initialPosition += step; //updating current position with each step
		}
		finalPositions[id] = initialPosition; //store the final position in the array
	}
}
/*
 RAND_MAX = 2147483647
 rand() returns a value in [0, 2147483647].
 Because RAND_MAX is odd and we are also using 0 this is an even number.
 Hence there is no middle interger so RAND_MAX/2 will divide the number in half if it is a float.
 You might could do this faster with a clever idea using ints but I'm going to use a float.
 Also I'm not sure how long the string of random numbers is. I'm sure it is longer than 10,000.
 Before you use this as a huge string check this out.
*/
// int getRandomDirection()
// {	
// 	int randomNumber = rand();
	
// 	if(randomNumber < MidPoint) return(-1);
// 	else return(1);
// }

int main(int argc, char** argv)
{
	int CPUfinalPositions[WALKS];
	int *GPUfinalPositions;

	int numberOfRandomSteps = STEPS;
	float MidPoint = (float)UINT_MAX / 2.0f; // Midpoint for random number generation

	hipMalloc(&GPUfinalPositions, WALKS * sizeof(int)); // Allocate memory on the GPU for final positions

	randomWalk<<<1, WALKS>>>(GPUfinalPositions, time(NULL), numberOfRandomSteps, MidPoint); // Launch kernel
	hipDeviceSynchronize();

	hipMemcpy(CPUfinalPositions, GPUfinalPositions, WALKS * sizeof(int), hipMemcpyDeviceToHost); // Copy results back to CPU

	//srand(time(NULL));
	
	printf(" Final positions of the random walks:\n");
	for(int i = 0; i < WALKS; i++)
	{
		printf("Walk %d: %d\n", i, CPUfinalPositions[i]); // Print final positions of each walk
	}
	
	hipFree(GPUfinalPositions); // Free GPU memory
	
	return 0;
}

