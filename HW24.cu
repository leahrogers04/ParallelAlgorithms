#include "hip/hip_runtime.h"
// Name:
// nBody code on multiple GPUs. 
// nvcc HW24.cu -o temp -lglut -lm -lGLU -lGL

/*
 What to do:
 This is some robust N-body code with all the bells and whistles removed. 
 Modify it so it runs on two GPUs.
*/

// Include files
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Defines
#define BLOCK_SIZE 128
#define PI 3.14159265359
#define DRAW_RATE 10

// This is to create a Lennard-Jones type function G/(r^p) - H(r^q). (p < q) p has to be less than q.
// In this code we will keep it a p = 2 and q = 4 problem. The diameter of a body is found using the general
// case so it will be more robust but in the code leaving it as a set 2, 4 problem make the coding much easier.
#define G 10.0f
#define H 10.0f
#define LJP  2.0
#define LJQ  4.0

#define DT 0.0001
#define RUN_TIME 10.0

// Globals
int N, N1, N2;
float3 *P, *V, *F;
float *M; 
float3 *PGPU1, *VGPU1, *FGPU1;
float3 *PGPU2, *VGPU2, *FGPU2;
float *MGPU1, *MGPU2;
float GlobeRadius, Diameter, Radius;
float Damp;
dim3 BlockSize;
dim3 GridSize1;
dim3 GridSize2;
//int halfN = (N+1)/2; // This is the half of the vector length
//int otherHalfN = N - halfN; // This is the other half of the vector length


// Function prototypes
void cudaErrorCheck(const char *, int);
void drawPicture();
void setup();
__global__ void getForces(float3 *, float3 *, float3 *, float *, float, float, int);
__global__ void moveBodies(float3 *, float3 *, float3 *, float *, float, float, float, int);
void nBody();
int main(int, char**);

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

void drawPicture()
{
	int i;
	
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	// hipSetDevice(0); // Set the first GPU
	// hipMemcpyAsync(PGPU1, P, N1*sizeof(float3), hipMemcpyHostToDevice);	
	// cudaErrorCheck(__FILE__, __LINE__);

	// hipSetDevice(1); // Set the second GPU
	// hipMemcpyAsync(PGPU2, P, N2*sizeof(float3), hipMemcpyDeviceToHost);
	// cudaErrorCheck(__FILE__, __LINE__);
	
	glColor3d(1.0,1.0,0.5);
	for(i=0; i<N; i++)
	{
		glPushMatrix();
		glTranslatef(P[i].x, P[i].y, P[i].z);
		glutSolidSphere(Radius,20,20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

void setup()
{
	int count = 0;
	hipGetDeviceCount(&count);
	if (count < 2)
	{
		printf("you only have %d 2 GPU's stupid idiot loser \n", count);
		exit (0);

	}
    	float randomAngle1, randomAngle2, randomRadius;
    	float d, dx, dy, dz;
    	int test;
    	
    	N = 1000;
		N1 = N/2; // This is the half of the vector length
		N2 = N - N1; // This is the other half of the vector length
		
		// float H = 10.0f;
		// float G = 10.0f;
		
		// float LJP = 2.0;
		// float LJQ = 4.0;
		
		float GlobeRadius, Diameter, Radius;
		
		// This is the layout of the parallel space we will be using.
		BlockSize.x = BLOCK_SIZE;
    	
    BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize1.x = (N1 - 1) / BlockSize.x + 1; //Makes enough blocks to deal with the whole vector.
	GridSize1.y = 1;
	GridSize1.z = 1;
	
	GridSize2.x = (N2 - 1) / BlockSize.x + 1; //Makes enough blocks to deal with the whole vector.
	GridSize2.y = 1;
	GridSize2.z = 1;
	

    	Damp = 0.5;
    	
    	M = (float*)malloc(N*sizeof(float));
    	P = (float3*)malloc(N*sizeof(float3));
    	V = (float3*)malloc(N*sizeof(float3));
    	F = (float3*)malloc(N*sizeof(float3));
    	
	hipSetDevice(0); // Set the first GPU
	hipMalloc(&PGPU1,N*sizeof(float3));			
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&VGPU1,N*sizeof(float3));	
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&FGPU1,N*sizeof(float3));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&MGPU1,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
    
	hipSetDevice(1); // Set the second GPU
	hipMalloc(&PGPU2,N*sizeof(float3));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&VGPU2,N*sizeof(float3));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&FGPU2,N*sizeof(float3));
	cudaErrorCheck(__FILE__, __LINE__);	
	hipMalloc(&MGPU2,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
    	
	Diameter = pow(H/G, 1.0/(LJQ - LJP)); // This is the value where the force is zero for the L-J type force.
	Radius = Diameter/2.0;
	
	// Using the radius of a body and a 68% packing ratio to find the radius of a global sphere that should hold all the bodies.
	// Then we double this radius just so we can get all the bodies setup with no problems. 
	float totalVolume = float(N)*(4.0/3.0)*PI*Radius*Radius*Radius;
	totalVolume /= 0.68;
	float totalRadius = pow(3.0*totalVolume/(4.0*PI), 1.0/3.0);
	GlobeRadius = 2.0*totalRadius;
	
	// Randomly setting these bodies in the glaobal sphere and setting the initial velosity, inotial force, and mass.
	for(int i = 0; i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			// Get random position.
			randomAngle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			randomAngle2 = ((float)rand()/(float)RAND_MAX)*PI;
			randomRadius = ((float)rand()/(float)RAND_MAX)*GlobeRadius;
			P[i].x = randomRadius*cos(randomAngle1)*sin(randomAngle2);
			P[i].y = randomRadius*sin(randomAngle1)*sin(randomAngle2);
			P[i].z = randomRadius*cos(randomAngle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = P[i].x-P[j].x;
				dy = P[i].y-P[j].y;
				dz = P[i].z-P[j].z;
				d = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < Diameter)
				{
					test = 0;
					break;
				}
			}
		}
	
		V[i].x = 0.0;
		V[i].y = 0.0;
		V[i].z = 0.0;
		
		F[i].x = 0.0;
		F[i].y = 0.0;
		F[i].z = 0.0;
		
		M[i] = 1.0;
	}
	hipSetDevice(0); // Set the first GPU
	hipMemcpyAsync(PGPU1, P, N1*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(VGPU1, V, N1*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(FGPU1, F, N1*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(MGPU1, M, N1*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);

	hipSetDevice(1); // Set the second GPU
	hipMemcpyAsync(PGPU2, P, N2*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(VGPU2, V, N2*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(FGPU2, F+N1, N2*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(MGPU2, M+N1, N2*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);

// 	int canAccessPeer01, canAccessPeer10;
// hipDeviceCanAccessPeer(&canAccessPeer01, 0, 1);
// hipDeviceCanAccessPeer(&canAccessPeer10, 1, 0);

// if (canAccessPeer01) {
//     hipSetDevice(0);
//     hipDeviceEnablePeerAccess(1, 0);
// }

// if (canAccessPeer10) {
//     hipSetDevice(1);
//     hipDeviceEnablePeerAccess(0, 0);
// }
 }

__global__ void getForces(float3 *p, float3 *v, float3 *f, float *m, float g, float h, int n)
{
	float dx, dy, dz,d,d2;
	float force_mag;
	
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	if(i < n)
	{
		f[i].x = 0.0f;
		f[i].y = 0.0f;
		f[i].z = 0.0f;
		
		for(int j = 0; j < n; j++)
		{
			if(i != j)
			{
				dx = p[j].x-p[i].x;
				dy = p[j].y-p[i].y;
				dz = p[j].z-p[i].z;
				d2 = dx*dx + dy*dy + dz*dz;
				d  = sqrt(d2);
				
				force_mag  = (g*m[i]*m[j])/(d2) - (h*m[i]*m[j])/(d2*d2);
				f[i].x += force_mag*dx/d;
				f[i].y += force_mag*dy/d;
				f[i].z += force_mag*dz/d;
			}
		}
	}
}

__global__ void moveBodies(float3 *p, float3 *v, float3 *f, float *m, float damp, float dt, float t, int n)
{	
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	if(i < n)
	{
		if(t == 0.0f)
		{
			v[i].x += ((f[i].x-damp*v[i].x)/m[i])*dt/2.0f;
			v[i].y += ((f[i].y-damp*v[i].y)/m[i])*dt/2.0f;
			v[i].z += ((f[i].z-damp*v[i].z)/m[i])*dt/2.0f;
		}
		else
		{
			v[i].x += ((f[i].x-damp*v[i].x)/m[i])*dt;
			v[i].y += ((f[i].y-damp*v[i].y)/m[i])*dt;
			v[i].z += ((f[i].z-damp*v[i].z)/m[i])*dt;
		}

		p[i].x += v[i].x*dt;
		p[i].y += v[i].y*dt;
		p[i].z += v[i].z*dt;
	}
}

void nBody()
{
	int    drawCount = 0; 
	float  t = 0.0;
	float dt = 0.0001;

	while(t < RUN_TIME)
	{
		hipSetDevice(0); // Set the first GPU
		getForces<<<GridSize1,BlockSize>>>(PGPU1, VGPU1, FGPU1, MGPU1, G, H, N1);
		cudaErrorCheck(__FILE__, __LINE__);
		moveBodies<<<GridSize1,BlockSize>>>(PGPU1, VGPU1, FGPU1, MGPU1, Damp, dt, t, N1);
		cudaErrorCheck(__FILE__, __LINE__);

		hipSetDevice(1); // Set the second GPU
		getForces<<<GridSize2,BlockSize>>>(PGPU2, VGPU2, FGPU2, MGPU2, G, H, N2);
		cudaErrorCheck(__FILE__, __LINE__);
		moveBodies<<<GridSize2,BlockSize>>>(PGPU2, VGPU2, FGPU2, MGPU2, Damp, dt, t, N2);
		cudaErrorCheck(__FILE__, __LINE__);

		hipDeviceSynchronize(); // Wait for the GPU to finish before moving on.

		hipSetDevice(0); // Set the first GPU	
		hipDeviceSynchronize(); // Wait for the GPU to finish before moving on.

		hipSetDevice(0); // Set the first GPU
		hipMemcpyAsync(PGPU2, PGPU1, N1*sizeof(float3), hipMemcpyDeviceToDevice);
		cudaErrorCheck(__FILE__, __LINE__);

		hipSetDevice(1); // Set the second GPU
		hipMemcpyAsync(PGPU1+N1, PGPU2+N1, N2*sizeof(float3), hipMemcpyDeviceToDevice);
		cudaErrorCheck(__FILE__, __LINE__);

		hipSetDevice(0); // Set the first GPU
		hipDeviceSynchronize(); // Wait for the GPU to finish before moving on.
		hipSetDevice(1); // Set the second GPU	
		hipDeviceSynchronize(); // Wait for the GPU to finish before moving on.
		if(drawCount == DRAW_RATE) 
		{	
			hipSetDevice(0); // Set the first GPU
			hipMemcpyAsync(P, PGPU1, N1*sizeof(float3), hipMemcpyDeviceToHost);
			cudaErrorCheck(__FILE__, __LINE__);

			hipSetDevice(1); // Set the second GPU
			hipMemcpyAsync(P+N1,PGPU2+N1, N2*sizeof(float3), hipMemcpyDeviceToHost);
			cudaErrorCheck(__FILE__, __LINE__);

			hipDeviceSynchronize(); // Wait for the GPU to finish before moving on.
			hipSetDevice(0); // Set the first GPU
			hipDeviceSynchronize(); // Wait for the GPU to finish before moving on.
			drawPicture();
			drawCount = 0;
		}
		
		t += dt;
		drawCount++;
	}
}

int main(int argc, char** argv)
{
	setup();
	
	int XWindowSize = 1000;
	int YWindowSize = 1000;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Nbody Two GPUs");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(drawPicture);
	glutIdleFunc(nBody);
	
	float3 eye = {0.0f, 0.0f, 2.0f*GlobeRadius};
	float near = 0.2;
	float far = 5.0*GlobeRadius;
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, near, far);
	glMatrixMode(GL_MODELVIEW);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	gluLookAt(eye.x, eye.y, eye.z, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	
	glutMainLoop();
	return 0;
}

