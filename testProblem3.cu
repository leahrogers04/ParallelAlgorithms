
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 1001 // Length of the vector
#define B 200
#define BLOCK_SIZE 10
// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
float DotCPU, DotGPU;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void innitialize();
void dotProductCPU(float*, float*, int);
__global__ void dotProductGPU(float*, float*, float*, int);
bool  check(float, float, float);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
    hipError_t  error;
    error = hipGetLastError();

    if(error != hipSuccess)
    {
        printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
        exit(0);
    }
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
   
    BlockSize.x = BLOCK_SIZE;
    BlockSize.y = 1;
    BlockSize.z = 1;
   
    GridSize.x = 4;
    GridSize.y = 1;
    GridSize.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{  
    // Host "CPU" memory.              
    A_CPU = (float*)malloc(N*sizeof(float));
    B_CPU = (float*)malloc(N*sizeof(float));
    C_CPU = (float*)malloc(N*sizeof(float));
   
    // Device "GPU" Memory
    hipMalloc(&A_GPU,N*sizeof(float));
    cudaErrorCheck(__FILE__, __LINE__);
    hipMalloc(&B_GPU,N*sizeof(float));
    cudaErrorCheck(__FILE__, __LINE__);
    hipMalloc(&C_GPU,N*sizeof(float));
    cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will add.
void innitialize()
{
    for(int i = 0; i < N; i++)
    {      
        A_CPU[i] = (float)i;    
        B_CPU[i] = (float)(3*i);
    }
}

// Adding vectors a and b on the CPU then stores result in vector c.
void dotProductCPU(float *a, float *b, float *C_CPU, int n)
{
    for(int id = 0; id < n; id++)
    {
        C_CPU[id] = a[id] * b[id];
    }
   
    for(int id = 1; id < n; id++)
    {
        C_CPU[0] += C_CPU[id];
    }
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void dotProductGPU(float *a, float *b, float *c, int n)
{
   
    int tid = threadIdx.x;
    __shared__ float s[BLOCK_SIZE];
    int id;
    int stride = blockDim.x*gridDim.x;
    int lastiteration = ((n-1)/stride)   +1;
    int fold;
    for (int i=0; i<=lastiteration;i++)
{
    id = tid+stride*i +blockDim.x*blockIdx.x;
    if (i<lastiteration)
    {
        s[tid] = a[id]*b[id];
   
    fold = blockDim.x;
    while(fold>1)
    {
        if(tid<fold/2)
        {
            s[tid] += s[fold-1-tid];
        }
        fold = (fold+1)/2;
    }
    }
    else if (i==lastiteration)
    {
        if(id<n)
        {
            s[tid] = a[id]*b[id];
        }
        else
        {
            s[tid] = 0;//padding with zeroes
        }
        fold = blockDim.x;
        while(fold>1)
        {
            if(tid<fold/2)
            {
                s[tid] += s[fold-1-tid];
            }
            fold = (fold+1)/2;
        }
    }
    if(tid==0)
    {
    atomicAdd(&c[0],s[tid]);
    }
}
}






// Checking to see if anything went wrong in the vector addition.
bool check(float cpuAnswer, float gpuAnswer, float tolerence)
{
    double percentError;
   
    percentError = abs((gpuAnswer - cpuAnswer)/(cpuAnswer))*100.0;
    printf("\n\n percent error = %lf\n", percentError);
   
    if(percentError < Tolerance)
    {
        return(true);
    }
    else
    {
        return(false);
    }
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
    // tv_sec = number of seconds past the Unix epoch 01/01/1970
    // tv_usec = number of microseconds past the current second.
   
    long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
    long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

    // Returning the total time elasped in microseconds
    return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
    // Freeing host "CPU" memory.
    free(A_CPU);
    free(B_CPU);
    free(C_CPU);
   
    hipFree(A_GPU);
    cudaErrorCheck(__FILE__, __LINE__);
    hipFree(B_GPU);
    cudaErrorCheck(__FILE__, __LINE__);
    hipFree(C_GPU);
    cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
    int DotGPU = 0;
    timeval start, end;
    long timeCPU, timeGPU;
    //float localC_CPU, localC_GPU;
   
    // Setting up the GPU
    setUpDevices();
   
    // Allocating the memory you will need.
    allocateMemory();
   
    // Putting values in the vectors.
    innitialize();
   
    // Adding on the CPU
    gettimeofday(&start, NULL);
    dotProductCPU(A_CPU, B_CPU, C_CPU, N);
    DotCPU = C_CPU[0];
    gettimeofday(&end, NULL);
    timeCPU = elaspedTime(start, end);
   
   
   
    // Adding on the GPU
    gettimeofday(&start, NULL);
   
    // Copy Memory from CPU to GPU      
    hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
    cudaErrorCheck(__FILE__, __LINE__);
    hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
    cudaErrorCheck(__FILE__, __LINE__);
   
    dotProductGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU, C_GPU, N);
    cudaErrorCheck(__FILE__, __LINE__);
   
    // Copy Memory from GPU to CPU  
    hipMemcpyAsync(C_CPU, C_GPU, sizeof(float), hipMemcpyDeviceToHost);
    cudaErrorCheck(__FILE__, __LINE__);
   
    DotGPU = C_CPU[0]; // C_GPU was copied into C_CPU.
   
    // Making sure the GPU and CPU wiat until each other are at the same place.
    hipDeviceSynchronize();
    cudaErrorCheck(__FILE__, __LINE__);

    gettimeofday(&end, NULL);
    timeGPU = elaspedTime(start, end);
   
    // Checking to see if all went correctly.
    if(check(DotCPU, DotGPU, Tolerance) == false)
    {
        printf("\n\n Something went wrong in the GPU dot product.\n");
    }
    else
    {
        printf("\n\n You did a dot product correctly on the GPU");
        printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
        printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
    }
   
    // Your done so cleanup your room.  
    CleanUp();  
   
    // Making sure it flushes out anything in the print buffer.
    printf("\n\n");
   
    return(0);
}
