// Name:Leah Rogers
// Page-locked memory test
// nvcc HW14.cu -o temp

/*
 What to do:
 Read chapter 10 in your book and file in the ???s in this code to understand how to setup and
 test page-locked memory on the host.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
//#include "./MyCuda.h"

// Defines
#define SIZE 2000000 
#define NUMBER_OF_COPIES 1000

//Globals
float *NumbersOnGPU, *PageableNumbersOnCPU, *PageLockedNumbersOnCPU;
hipEvent_t StartEvent, StopEvent;

//Function prototypes
void cudaErrorCheck(const char *, int);
void setUpCudaDevices();
void allocateMemory();
void cleanUp();
void copyPageableMemoryUp();
void copyPageLockedMemoryUp();
void copyPageableMemoryDown();
void copyPageLockedMemoryDown();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

//This will be the layout of the parallel space we will be using.
void setUpCudaDevices()
{
	hipEventCreate(&StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventCreate(&StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
}

//Sets a side memory on the GPU and CPU for our use.
void allocateMemory()
{					
	//Allocate Device (GPU) Memory
	hipMalloc(&NumbersOnGPU, SIZE*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);

	//Allocate pageable Host (CPU) Memory
	PageableNumbersOnCPU = (float*)malloc(SIZE*sizeof(float));
	
	//Allocate page locked Host (CPU) Memory
	//???
	hipHostAlloc(&PageLockedNumbersOnCPU, SIZE*sizeof(float), hipHostMallocDefault);
	cudaErrorCheck(__FILE__, __LINE__);
}

//Cleaning up memory after we are finished.
void cleanUp()
{
	hipFree(NumbersOnGPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	
	//?????
	hipHostFree(PageLockedNumbersOnCPU);
	cudaErrorCheck(__FILE__, __LINE__);
	
	free(PageableNumbersOnCPU); 
	
	hipEventDestroy(StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventDestroy(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
}

void copyPageableMemoryUp()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(NumbersOnGPU, PageableNumbersOnCPU, SIZE*sizeof(float), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageableMemoryDown()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(PageableNumbersOnCPU, NumbersOnGPU, SIZE*sizeof(float), hipMemcpyDeviceToHost);
		cudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageLockedMemoryUp()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		//???
		hipMemcpy(NumbersOnGPU, PageLockedNumbersOnCPU, SIZE*sizeof(float), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageLockedMemoryDown()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		//?????
		hipMemcpy(PageLockedNumbersOnCPU, NumbersOnGPU, SIZE*sizeof(float), hipMemcpyDeviceToHost);
		cudaErrorCheck(__FILE__, __LINE__);
	}
}


int main()
{
	float timeEvent;
	
	setUpCudaDevices();
	allocateMemory();
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageableMemoryUp();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using pageable memory up = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageLockedMemoryUp();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using page locked memory up = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageableMemoryDown();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using pageable memory down = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageLockedMemoryDown();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using page locked memory down = %3.1f milliseconds", timeEvent);
	
	printf("\n");
	//You're done so cleanup your mess.
	cleanUp();	
	
	return(0);
}
