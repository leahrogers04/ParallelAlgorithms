// Name: Leah Rogers
// Vector addition on two GPUs.
// nvcc HW22.cu -o temp
/*
 What to do:
 This code adds two vectors of any length on a GPU.
 Rewriting the Code to Run on Two GPUs:

 1. Check GPU Availability:
    Ensure that you have at least two GPUs available. If not, report the issue and exit the program.

 2. Handle Odd-Length Vector:
    If the vector length is odd, ensure that you select a half N value that does not exclude the last element of the vector.

 3. Send First Half to GPU 1:
    Send the first half of the vector to the first GPU, and perform the operation of adding a to b.

 4. Send Second Half to GPU 2:
    Send the second half of the vector to the second GPU, and again perform the operation of adding a to b.

 5. Return Results to the CPU:
    Once both GPUs have completed their computations, transfer the results back to the CPU and verify that the results are correct.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 11503 // Length of the vector

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU1, *B_GPU1, *C_GPU1; //GPU pointers
float *A_GPU2, *B_GPU2, *C_GPU2; //GPU pointers
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize1;
dim3 GridSize2; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;
int halfN = (N+1)/2; // This is the half of the vector length
int otherHalfN = N - halfN; // This is the other half of the vector length

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void innitialize();
void addVectorsCPU(float*, float*, float*, int);
__global__ void addVectorsGPU(float, float, float, int);
bool  check(float*, int);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	int count = 0;
	hipGetDeviceCount(&count);
	if (count < 2)
	{
		printf("you need 2 GPUs and you only have %d\n", count);
		exit (1);
	}




	BlockSize.x = 256;
	BlockSize.y = 1;
	BlockSize.z = 1;
	

	GridSize1.x = (halfN + BlockSize.x - 1) / BlockSize.x;
	GridSize1.y = 1;
	GridSize1.z = 1;

	GridSize2.x = (otherHalfN + BlockSize.x - 1) / BlockSize.x;
	GridSize2.y = 1;
	GridSize2.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipSetDevice(0); // Set the first GPU
	hipMalloc(&A_GPU1,halfN*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU1,halfN*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU1,halfN*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);

	hipSetDevice(1); // Set the second GPU
	hipMalloc(&A_GPU2,otherHalfN*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU2,otherHalfN*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU2,otherHalfN*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(2*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void addVectorsCPU(float *a, float *b, float *c, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		c[id] = a[id] + b[id];
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void addVectorsGPU(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id < n) // Making sure we are not working on memory we do not own.
	{
		c[id] = a[id] + b[id];
	}
}

// Checking to see if anything went wrong in the vector addition.
bool check(float *c, int n, float tolerence)
{
	int id;
	double myAnswer;
	double trueAnswer;
	double percentError;
	double m = n-1; // Needed the -1 because we start at 0.
	
	myAnswer = 0.0;
	for(id = 0; id < n; id++)
	{ 
		myAnswer += c[id];
	}
	
	trueAnswer = 3.0*(m*(m+1))/2.0;
	
	percentError = abs((myAnswer - trueAnswer)/trueAnswer)*100.0;
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU1); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU1); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU1);
	cudaErrorCheck(__FILE__, __LINE__);

	hipFree(A_GPU2);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU2);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU2);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	addVectorsCPU(A_CPU, B_CPU ,C_CPU, N);
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	// Zeroing out the C_CPU vector just to be safe because right now it has the correct answer in it.
	for(int id = 0; id < N; id++)
	{ 
		C_CPU[id] = 0.0;
	}
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU1, A_CPU, halfN*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU1, B_CPU, halfN*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	addVectorsGPU<<<GridSize1,BlockSize>>>(A_GPU1, B_GPU1 ,C_GPU1, halfN);
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU1, halfN*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipMemcpyAsync(A_GPU2, A_CPU + halfN, otherHalfN*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU2, B_CPU + halfN, otherHalfN*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	addVectorsGPU<<<GridSize2,BlockSize>>>(A_GPU2, B_GPU2 ,C_GPU2, otherHalfN);
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);

	hipMemcpyAsync(C_CPU + halfN, C_GPU2, otherHalfN*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	// Making sure the GPU and CPU wiat until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);
	
	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Checking to see if all went correctly.
	if(check(C_CPU, N, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU vector addition\n");
	}
	else
	{
		printf("\n\n You added the two vectors correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}

