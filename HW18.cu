#include "hip/hip_runtime.h"
// Name: 
// Creating a GPU nBody simulation from an nBody CPU simulation. 
// nvcc HW18.cu -o temp -lglut -lm -lGLU -lGL

/*
 What to do:
 This is some lean nBody code that runs on the CPU. Rewrite it, keeping the same general format, 
 but offload the compute-intensive parts of the code to the GPU for acceleration.
 Note: The code takes two arguments as inputs:
 1. The number of bodies to simulate, (We will keep the number of bodies under 1024 for this HW so it can be run on one block.)
 2. Whether to draw sub-arrangements of the bodies during the simulation (1), or only the first and last arrangements (0).
*/

// Include files
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Defines
#define PI 3.14159265359
#define DRAW_RATE 10

// This is to create a Lennard-Jones type function G/(r^p) - H(r^q). (p < q) p has to be less than q.
// In this code we will keep it a p = 2 and q = 4 problem. The diameter of a body is found using the general
// case so it will be more robust but in the code leaving it as a set 2, 4 problem make the coding much easier.
#define G 10.0
#define H 10.0
#define LJP  2.0
#define LJQ  4.0

#define DT 0.0001
#define RUN_TIME 1.0

// Globals
int N, DrawFlag;
float3 *P, *V, *F;
float *M; 
float3 *P_GPU, *V_GPU, *F_GPU;
float *M_GPU;
float GlobeRadius, Diameter, Radius;
float Damp;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid

//__constant__ float Damp_GPU;

// Function prototypes
void KeyPressed(unsigned char, int, int);
long elaspedTime(struct timeval, struct timeval);
void drawPicture();
void timer();
void setup();
void nBody();
void __global__ forcesGPU(float3 *P, float3 *V, float3 *F, float *M, int N);
void __global__ positionsGPU(float3 *P, float3 *V, float3 *F, float *M, int N, float Damp_GPU);
int main(int, char**);
void cleanUp();

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

void setUpDevices()
{
	BlockSize.x = 1024;
	BlockSize.y = 1;
	BlockSize.z = 1;

	
	GridSize.x = 1;
	GridSize.y = 1;
	GridSize.z = 1;

}


void KeyPressed(unsigned char key, int x, int y)
{
	if(key == 's')
	{
		timer();
	}
	
	if(key == 'q')
	{
		exit(0);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

void drawPicture()
{
	int i;
	
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(1.0,1.0,0.5);
	for(i=0; i<N; i++)
	{
		glPushMatrix();
		glTranslatef(P[i].x, P[i].y, P[i].z);
		glutSolidSphere(Radius,20,20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

void timer()
{	
	timeval start, end;
	long computeTime;
	
	drawPicture();
	gettimeofday(&start, NULL);
    		nBody();
    	gettimeofday(&end, NULL);
    	drawPicture();
    	
	computeTime = elaspedTime(start, end);
	printf("\n The compute time was %ld microseconds.\n\n", computeTime);
}

void setup()
{
    	float randomAngle1, randomAngle2, randomRadius;
    	float d, dx, dy, dz;
    	int test;
    	
    	Damp = 0.5;
    	
    	M = (float*)malloc(N*sizeof(float));
    	P = (float3*)malloc(N*sizeof(float3));
    	V = (float3*)malloc(N*sizeof(float3));
    	F = (float3*)malloc(N*sizeof(float3));
    	
		//memory allocation for the GPU
		hipMalloc(&F_GPU, N*sizeof(float3));
		cudaErrorCheck(__FILE__, __LINE__);

		hipMalloc(&P_GPU, N*sizeof(float3));
		cudaErrorCheck(__FILE__, __LINE__);

		hipMalloc(&V_GPU, N*sizeof(float3));
		cudaErrorCheck(__FILE__, __LINE__);

		hipMalloc(&M_GPU, N*sizeof(float));
		cudaErrorCheck(__FILE__, __LINE__);
		
		// Setting the diameter of the bodies.
	
	Diameter = pow(H/G, 1.0/(LJQ - LJP)); // This is the value where the force is zero for the L-J type force.
	Radius = Diameter/2.0;
	
	// Using the radius of a body and a 68% packing ratio to find the radius of a global sphere that should hold all the bodies.
	// Then we double this radius just so we can get all the bodies setup with no problems. 
	float totalVolume = float(N)*(4.0/3.0)*PI*Radius*Radius*Radius;
	totalVolume /= 0.68;
	float totalRadius = pow(3.0*totalVolume/(4.0*PI), 1.0/3.0);
	GlobeRadius = 2.0*totalRadius;
	
	// Randomly setting these bodies in the glaobal sphere and setting the initial velosity, inotial force, and mass.
	for(int i = 0; i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			// Get random position.
			randomAngle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			randomAngle2 = ((float)rand()/(float)RAND_MAX)*PI;
			randomRadius = ((float)rand()/(float)RAND_MAX)*GlobeRadius;
			P[i].x = randomRadius*cos(randomAngle1)*sin(randomAngle2);
			P[i].y = randomRadius*sin(randomAngle1)*sin(randomAngle2);
			P[i].z = randomRadius*cos(randomAngle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = P[i].x-P[j].x;
				dy = P[i].y-P[j].y;
				dz = P[i].z-P[j].z;
				d = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < Diameter)
				{
					test = 0;
					break;
				}
			}
		}
	
		V[i].x = 0.0;
		V[i].y = 0.0;
		V[i].z = 0.0;
		
		F[i].x = 0.0;
		F[i].y = 0.0;
		F[i].z = 0.0;
		
		M[i] = 1.0;
	}
	printf("\n To start timing type s.\n");
}

void nBody()
{
	// This is the CPU version of the nBody code.

	hipMemcpy(P_GPU, P, N*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpy(V_GPU, V, N*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpy(F_GPU, F, N*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpy(M_GPU, M, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	//hipMemcpyToSymbol(HIP_SYMBOL(Damp_GPU), &Damp, sizeof(float));
	//cudaErrorCheck(__FILE__, __LINE__);


	// float force_mag; 
	// float dx,dy,dz,d, d2;
	
	int    drawCount = 0; 
	float  time = 0.0;
	float dt = 0.0001;

	while(time < RUN_TIME)
	{
		forcesGPU<<<GridSize, BlockSize>>>(P_GPU, V_GPU, F_GPU, M_GPU, N);
		cudaErrorCheck(__FILE__, __LINE__);

		positionsGPU<<<GridSize, BlockSize>>>(P_GPU, V_GPU, F_GPU, M_GPU, N, Damp);
		cudaErrorCheck(__FILE__, __LINE__);
		
		hipMemcpy(P, P_GPU, N*sizeof(float3), hipMemcpyDeviceToHost);
		cudaErrorCheck(__FILE__, __LINE__);
		// for(int i=0; i<N; i++)
		// {
		// 	for(int j=i+1; j<N; j++)
		// 	{
		// 		dx = P[j].x-P[i].x;
		// 		dy = P[j].y-P[i].y;
		// 		dz = P[j].z-P[i].z;
		// 		d2 = dx*dx + dy*dy + dz*dz;
		// 		d  = sqrt(d2);
				
		// 		force_mag  = (G*M[i]*M[j])/(d2) - (H*M[i]*M[j])/(d2*d2);
		// 		F[i].x += force_mag*dx/d;
		// 		F[j].x -= force_mag*dx/d;
		// 		F[i].y += force_mag*dy/d;
		// 		F[j].y -= force_mag*dy/d;
		// 		F[i].z += force_mag*dz/d;
		// 		F[j].z -= force_mag*dz/d;
		// 	}
		// }

		// for(int i=0; i<N; i++)
		// {
		// 	if(time == 0.0)
		// 	{
		// 		V[i].x += (F[i].x/M[i])*0.5*dt;
		// 		V[i].y += (F[i].y/M[i])*0.5*dt;
		// 		V[i].z += (F[i].z/M[i])*0.5*dt;
		// 	}
		// 	else
		// 	{
		// 		V[i].x += ((F[i].x-Damp*V[i].x)/M[i])*dt;
		// 		V[i].y += ((F[i].y-Damp*V[i].y)/M[i])*dt;
		// 		V[i].z += ((F[i].z-Damp*V[i].z)/M[i])*dt;
		// 	}

		// 	P[i].x += V[i].x*dt;
		// 	P[i].y += V[i].y*dt;
		// 	P[i].z += V[i].z*dt;
		// }

		if(drawCount == DRAW_RATE) 
		{
			if(DrawFlag) drawPicture();
			drawCount = 0;
		}
		
		time += dt;
		drawCount++;
	}
}



void __global__ forcesGPU(float3 *P_GPU, float3 *V_GPU, float3 *F_GPU, float *M_GPU, int N)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	float force_mag;
	float dx,dy,dz,d, d2;
	

	if(id >= N) return; // this makes sure that the threads outside the range of bodies do not do any work.

	// set the initial forces to zero
	F_GPU[id].x = 0.0;
	F_GPU[id].y = 0.0;
	F_GPU[id].z = 0.0;

	for(int i=0; i<N; i++)
		{
			for(int j=i+1; j<N; j++)
			{
				dx = P_GPU[j].x-P_GPU[i].x;
				dy = P_GPU[j].y-P_GPU[i].y;
				dz = P_GPU[j].z-P_GPU[i].z;
				d2 = dx*dx + dy*dy + dz*dz;
				d  = sqrt(d2);
				
				force_mag  = (G*M_GPU[i]*M_GPU[j])/(d2) - (H*M_GPU[i]*M_GPU[j])/(d2*d2);
				F_GPU[i].x += force_mag*dx/d;
				F_GPU[j].x -= force_mag*dx/d;
				F_GPU[i].y += force_mag*dy/d;
				F_GPU[j].y -= force_mag*dy/d;
				F_GPU[i].z += force_mag*dz/d;
				F_GPU[j].z -= force_mag*dz/d;
			}
		}

		



}

void __global__ positionsGPU(float3 *P_GPU, float3 *V_GPU, float3 *F_GPU, float *M_GPU, int N, float Damp_GPU)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	float dt = 0.0001;
	float time = 0.0;
	
	if (id >= N) return; // this makes sure that the threads outside the range of bodies do not do any work.
			if(time == 0.0)
			{
				V_GPU[id].x += (F_GPU[id].x/M_GPU[id])*0.5*dt;
				V_GPU[id].y += (F_GPU[id].y/M_GPU[id])*0.5*dt;
				V_GPU[id].z += (F_GPU[id].z/M_GPU[id])*0.5*dt;
				
			}
			else
			{
				V_GPU[id].x += ((F_GPU[id].x-Damp_GPU*V_GPU[id].x)/M_GPU[id])*dt;
				V_GPU[id].y += ((F_GPU[id].y-Damp_GPU*V_GPU[id].y)/M_GPU[id])*dt;
				V_GPU[id].z += ((F_GPU[id].z-Damp_GPU*V_GPU[id].z)/M_GPU[id])*dt;
				
			}

			P_GPU[id].x += V_GPU[id].x*dt;
			P_GPU[id].y += V_GPU[id].y*dt;	
			P_GPU[id].z += V_GPU[id].z*dt;
			
		
}

void cleanUp()
{
	free(P);
	free(V);
	free(F);
	free(M);

	hipFree(P_GPU);
	hipFree(V_GPU);
	hipFree(F_GPU);
	hipFree(M_GPU);

	printf("\n Memory has been cleaned up.\n");
}

int main(int argc, char** argv)
{
	if( argc < 3)
	{
		printf("\n You need to intire the number of bodies (an int)"); 
		printf("\n and if you want to draw the bodies as they move (1 draw, 0 don't draw),");
		printf("\n on the comand line.\n"); 
		exit(0);
	}
	else
	{
		N = atoi(argv[1]);
		DrawFlag = atoi(argv[2]);
	}
	
	setUpDevices();
	setup();
	
	int XWindowSize = 1000;
	int YWindowSize = 1000;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("nBody Test");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutKeyboardFunc(KeyPressed);
	
	float3 eye = {0.0f, 0.0f, 2.0f*GlobeRadius};
	float near = 0.2;
	float far = 5.0*GlobeRadius;
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, near, far);
	glMatrixMode(GL_MODELVIEW);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	gluLookAt(eye.x, eye.y, eye.z, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	
	glutMainLoop();
	return 0;
}





